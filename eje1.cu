
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel para sumar elementos de dos vectores
__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    // Asegúrate de no salir del rango de los vectores
    if (index < n) {
        c[index] = a[index] + b[index];
        printf("Thread #%d, sumando %d + %d = %d\n", index, a[index], b[index], c[index]);
    }
}

// Función principal
int main() {
    int n = 10; // Por ejemplo, tamaño de los vectores
    int *a, *b, *c; // vectores en el host
    int *d_a, *d_b, *d_c; // vectores en el device
    int size = n * sizeof(int);

    // Asignación de memoria en el host
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    // Inicializar los vectores de entrada con valores
    for(int i = 0; i < n; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Asignación de memoria en el device
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copiar los vectores de entrada al device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Lanzar el kernel en el device con un thread por cada elemento
    vectorAdd<<<(n+255)/256, 256>>>(d_a, d_b, d_c, n);

    // Copiar el vector de resultado de vuelta al host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Limpieza
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(a);
    free(b);
    free(c);

    // Verificación de errores de CUDA
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
        fprintf(stderr, "ERROR: %s\n", hipGetErrorString(error));
        return -1;
    }

    // Finalización exitosa
    return 0;
}
