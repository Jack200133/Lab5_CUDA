#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float *a, const float *b, float *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    int n = 1000000; // Tamaño grande para medir tiempo significativo
    size_t size = n * sizeof(float);

    float *a, *b, *c;
    float *d_a, *d_b, *d_c;

    a = (float *)malloc(size);
    b = (float *)malloc(size);
    c = (float *)malloc(size);

    for (int i = 0; i < n; i++) {
        a[i] = (float)rand() / (float)RAND_MAX;
        b[i] = (float)rand() / (float)RAND_MAX;
    }

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    vectorAdd<<<(n+255)/256, 256>>>(d_a, d_b, d_c, n);
    hipEventRecord(stop);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Calcular e imprimir la suma de todos los valores del vector resultante
    float sum = 0.0f;
    for (int i = 0; i < n; i++) {
        sum += c[i];
    }
    printf("Suma de todos los valores en C: %f\n", sum);


    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Tiempo de ejecución de CUDA: %f milisegundos\n", milliseconds);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
